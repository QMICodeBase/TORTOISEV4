#include "hip/hip_runtime.h"
#ifndef _QUADRATICTRANSFORMIMAGE_CUDA_CU
#define _QUADRATICTRANSFORMIMAGE_CUDA_CU

#include <stdio.h>
#include <iostream>

#include "cuda_utils.h"



#define BLOCKSIZE 32
#define PER_SLICE 1



__global__ void
RigidTransformImage_kernel(hipTextureObject_t tex, int3 target_sz, int3 img_sz,
                  float *dsmat, float *dsmat_inv,
                  float *drotmat, float *dparams, 
                  hipPitchedPtr output )
{
    uint i = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    uint j = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
    uint kk = __umul24(blockIdx.z, blockDim.z) + threadIdx.z;

    for(int k=PER_SLICE*kk;k<PER_SLICE*kk+PER_SLICE;k++)
    {
        if(i<target_sz.x && j <target_sz.y && k<target_sz.z)
        {
            size_t opitch= output.pitch;
            size_t oslicePitch= opitch*target_sz.y*k;
            size_t ocolPitch= j*opitch;

            char *o_ptr= (char *)(output.ptr);
            char * slice_o= o_ptr+  oslicePitch;
            float * row_out= (float *)(slice_o+ ocolPitch);

            float x= dsmat[0]*i + dsmat[1]*j + dsmat[2]*k+ dsmat[3];
            float y= dsmat[4]*i + dsmat[5]*j + dsmat[6]*k+ dsmat[7];
            float z= dsmat[8]*i + dsmat[9]*j + dsmat[10]*k+ dsmat[11];
               

            float x1= drotmat[0]*x + drotmat[1]*y + drotmat[2]*z + dparams[3];
            float y1= drotmat[3]*x + drotmat[4]*y + drotmat[5]*z + dparams[4];
            float z1= drotmat[6]*x + drotmat[7]*y + drotmat[8]*z + dparams[5];
                
            float iw= dsmat_inv[0]*x1 + dsmat_inv[1]*y1 + dsmat_inv[2]*z1 + dsmat_inv[3];
            float jw= dsmat_inv[4]*x1 + dsmat_inv[5]*y1 + dsmat_inv[6]*z1 + dsmat_inv[7];
            float kw= dsmat_inv[8]*x1 + dsmat_inv[9]*y1 + dsmat_inv[10]*z1 + dsmat_inv[11];


            if(iw>=0 && iw<=img_sz.x-1 && jw>=0 && jw<=img_sz.y-1 && kw>=0 && kw<=img_sz.z-1 )
                row_out[i] =tex3D<float>(tex, iw+0.5, jw +0.5, kw+0.5);
        }
    }
}


void RigidTransformImage_cuda(hipTextureObject_t tex,
                                  int3 img_sz,float3 img_res, float3 img_orig, float *img_dir,
                                  int3 target_sz,float3 target_res, float3 target_orig, float *target_dir,
                                  float rotmat_arr[],
                                  float params_arr[],
                                  hipPitchedPtr output )
{
    const dim3 blockSize(BLOCKSIZE, BLOCKSIZE, BLOCKSIZE);
    const dim3 gridSize(std::ceil(1.*target_sz.x / blockSize.x), std::ceil(1.*target_sz.y / blockSize.y), std::ceil(1.*target_sz.z / blockSize.z/PER_SLICE) );

    float smat[16]={0},smat_inv[16]={0};
    smat[0]= target_dir[0]*target_res.x;
    smat[1]= target_dir[1]*target_res.y;
    smat[2]= target_dir[2]*target_res.z;
    smat[4]= target_dir[3]*target_res.x;
    smat[5]= target_dir[4]*target_res.y;
    smat[6]= target_dir[5]*target_res.z;
    smat[8]= target_dir[6]*target_res.x;
    smat[9]= target_dir[7]*target_res.y;
    smat[10]=target_dir[8]*target_res.z;
    smat[3]= target_orig.x;
    smat[7]= target_orig.y;
    smat[11]= target_orig.z;
    smat[15]=1;    
    
    smat_inv[0]= img_dir[0]/img_res.x;
    smat_inv[1]= img_dir[3]/img_res.x;
    smat_inv[2]= img_dir[6]/img_res.x;
    smat_inv[4]= img_dir[1]/img_res.y;
    smat_inv[5]= img_dir[4]/img_res.y;
    smat_inv[6]= img_dir[7]/img_res.y;
    smat_inv[8]= img_dir[2]/img_res.z;
    smat_inv[9]= img_dir[5]/img_res.z;
    smat_inv[10]=img_dir[8]/img_res.z;    
    smat_inv[3]= -(smat_inv[0] *img_orig.x +smat_inv[1] *img_orig.y + smat_inv[2] *img_orig.z );
    smat_inv[7]= -(smat_inv[4] *img_orig.x +smat_inv[5] *img_orig.y + smat_inv[6] *img_orig.z );
    smat_inv[11]=-(smat_inv[8] *img_orig.x +smat_inv[9] *img_orig.y + smat_inv[10] *img_orig.z );
    smat_inv[15]=1;
                        

    float *dsmat,*dsmat_inv;
    hipMalloc((void**)&dsmat, sizeof(float)*16);
    hipMalloc((void**)&dsmat_inv, sizeof(float)*16);    
    hipMemcpy(dsmat, smat, sizeof(float)*16, hipMemcpyHostToDevice);
    hipMemcpy(dsmat_inv, smat_inv, sizeof(float)*16, hipMemcpyHostToDevice);
        
    float *d_rotmat,*d_params;
    hipMalloc((void**)&d_rotmat, sizeof(float)*9);
    hipMalloc((void**)&d_params, sizeof(float)*24);    

    hipMemcpy(d_rotmat, rotmat_arr, sizeof(float)*9, hipMemcpyHostToDevice);
    hipMemcpy(d_params, params_arr, sizeof(float)*24, hipMemcpyHostToDevice);
    
    RigidTransformImage_kernel<<< blockSize,gridSize>>>( tex,
                                                target_sz, img_sz,
                                                dsmat, dsmat_inv,
                                                d_rotmat,d_params,
                                                output );
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    
    hipFree(dsmat);
    hipFree(dsmat_inv);
    hipFree(d_rotmat);
    hipFree(d_params);
}


#endif
